#include "hip/hip_runtime.h"
#include "func.h"
#include <stdio.h>

__global__ void add(int n, int* c, int* a, int* b) {

    for (int i = blockIdx.x; i < n; i += gridDim.x) {
        c[i] = a[i] + b[i];
    }
    c[0] = blockDim.x;
    return;
}

void parrel_add(int n, int* a, int* b, int* c) {
    hipError_t er;

    //����gpu�ڴ�
    int* da = nullptr;
    int* db = nullptr;
    int* dc = nullptr;
    er = hipMalloc(&da, n * sizeof(int));
    assert(er == hipSuccess);
    er = hipMalloc(&db, n * sizeof(int));
    er = hipMalloc(&dc, n * sizeof(int));

    //�������ݵ�gpu
    er = hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);


    er = hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);

    //����gpu���м���
    add << <20, 1 >> > (n, dc, da, db);

    //���������cpu
    HANDLE_ERROR(hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost));

    //�ͷ�gpu
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return;

}



